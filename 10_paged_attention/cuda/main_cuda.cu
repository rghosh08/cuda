#include "hip/hip_runtime.h"
// main_cuda.cu
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>

#include "paged_attention_config.h"
#include "paged_flash_attention_cuda.h"

// Function to detect system capabilities
void detect_system_capabilities() {
    std::cout << "=== Fixed Paged Attention CUDA Implementation ===" << std::endl;
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    std::cout << "System Information:" << std::endl;
    std::cout << "  CUDA Devices: " << deviceCount << std::endl;
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        
        std::cout << "  Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "    Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "    Total Global Memory: " << deviceProp.totalGlobalMem / (1024*1024*1024) << " GB" << std::endl;
        std::cout << "    Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "    Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "    Max Shared Memory per Block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
    }
    
    std::cout << std::endl;
}

// Function to generate test data
void generate_test_data(
    int batch_size,
    int num_heads,
    int head_dim,
    int max_seq_len,
    std::vector<int>& seq_lengths,
    std::vector<float>& keys,
    std::vector<float>& values,
    std::vector<float>& queries
) {
    std::cout << "Generating test data with pattern: full context" << std::endl;
    
    // Set all sequence lengths to max_seq_len
    seq_lengths.resize(batch_size, max_seq_len);
    std::cout << "  Sequence lengths: ";
    for (int i = 0; i < batch_size; i++) {
        std::cout << seq_lengths[i] << " ";
    }
    std::cout << std::endl;
    
    // Calculate total tokens
    int total_tokens = 0;
    for (int len : seq_lengths) {
        total_tokens += len;
    }
    
    // Generate random data
    std::mt19937 rng(42);  // Fixed seed for reproducibility
    std::uniform_real_distribution<float> dist(-0.1f, 0.1f);
    
    // Resize and fill data
    keys.resize(total_tokens * num_heads * head_dim);
    values.resize(total_tokens * num_heads * head_dim);
    queries.resize(batch_size * num_heads * head_dim);
    
    for (size_t i = 0; i < keys.size(); i++) {
        keys[i] = dist(rng);
    }
    
    for (size_t i = 0; i < values.size(); i++) {
        values[i] = dist(rng);
    }
    
    for (size_t i = 0; i < queries.size(); i++) {
        queries[i] = dist(rng);
    }
}

// Run the test
void run_test() {
    std::cout << "🧪 Running Simple Test 🧪" << std::endl << std::endl;
    
    // Create and initialize PagedFlashAttention
    paged_attention::PagedFlashAttentionCUDA attention;
    
    // Generate test data
    std::vector<int> seq_lengths;
    std::vector<float> keys, values, queries;
    generate_test_data(
        attention.get_config().batch_size,
        attention.get_config().num_heads,
        attention.get_config().head_dim,
        attention.get_config().max_seq_len,
        seq_lengths,
        keys,
        values,
        queries
    );
    
    // Update KV cache for each sequence
    int key_offset = 0;
    for (int seq_idx = 0; seq_idx < attention.get_config().batch_size; seq_idx++) {
        int seq_len = seq_lengths[seq_idx];
        
        if (!attention.update_kv_cache(
            seq_idx,
            seq_len,
            keys.data() + key_offset,
            values.data() + key_offset
        )) {
            std::cerr << "Failed to update KV cache for sequence " << seq_idx << std::endl;
            return;
        }
        
        key_offset += seq_len * attention.get_config().num_heads * attention.get_config().head_dim;
    }
    
    // Print KV cache usage
    int total_pages_used = attention.get_total_pages_used();
    int total_pages = attention.get_config().num_pages;
    std::cout << "KV cache usage: " << total_pages_used << " / " << total_pages << " pages used ("
              << (100.0 * total_pages_used / total_pages) << "%)" << std::endl;
    
    // Allocate output buffer
    std::vector<float> outputs(attention.get_config().batch_size * 
                             attention.get_config().num_heads * 
                             attention.get_config().head_dim, 0.0f);
    
    // Compute attention
    bool success = attention.compute_attention(
        queries.data(),
        outputs.data(),
        seq_lengths.data(),
        1.0f / std::sqrt(attention.get_config().head_dim)
    );
    
    if (success) {
        // Check for NaN/Inf in outputs
        bool valid_output = true;
        for (float val : outputs) {
            if (std::isnan(val) || std::isinf(val)) {
                valid_output = false;
                break;
            }
        }
        
        if (valid_output) {
            std::cout << "✅ Test completed successfully with valid outputs!" << std::endl;
            
            // Print some sample outputs
            std::cout << "Sample outputs:" << std::endl;
            for (int i = 0; i < std::min(5, attention.get_config().head_dim); i++) {
                std::cout << "  output[0][0][" << i << "] = " << outputs[i] << std::endl;
            }
        } else {
            std::cout << "❌ Test failed: Output contains NaN or Inf values" << std::endl;
        }
    } else {
        std::cout << "❌ Test failed: Attention computation failed" << std::endl;
    }
}

// Function to print the configuration
void print_config(const paged_attention::PagedAttentionConfig& config) {
    std::cout << "PagedAttentionConfig:" << std::endl;
    std::cout << "  batch_size: " << config.batch_size << std::endl;
    std::cout << "  num_heads: " << config.num_heads << std::endl;
    std::cout << "  head_dim: " << config.head_dim << std::endl;
    std::cout << "  max_seq_len: " << config.max_seq_len << std::endl;
    std::cout << "  page_size: " << config.page_size << std::endl;
    std::cout << "  num_pages: " << config.num_pages << std::endl;
    std::cout << "  recycle_pages: " << (config.recycle_pages ? "true" : "false") << std::endl;
}

// Main function
int main() {
    auto start = std::chrono::high_resolution_clock::now();

    // Detect system capabilities
    detect_system_capabilities();
    
    // Print configuration
    print_config(paged_attention::DEFAULT_CONFIG);
    
    // Run the test
    run_test();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Total execution time: " << elapsed.count() << " seconds" << std::endl;
    
    return 0;
}
