#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS 256
#define BLOCKS 256
#define TOTAL_POINTS (THREADS * BLOCKS * 1000)

// GPU kernel: Monte Carlo simulation
__global__ void monte_carlo_pi(int *count) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(1234, id, 0, &state);

    int local_count = 0;
    for (int i = 0; i < 1000; i++) {
        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        if (x*x + y*y <= 1.0f) local_count++;
    }
    count[id] = local_count;
}

int main() {
    int *d_count, h_count[THREADS * BLOCKS];

    hipMalloc(&d_count, THREADS * BLOCKS * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    monte_carlo_pi<<<BLOCKS, THREADS>>>(d_count);
    hipDeviceSynchronize();

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_count, d_count, THREADS * BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

    int total_in_circle = 0;
    for (int i = 0; i < THREADS * BLOCKS; i++)
        total_in_circle += h_count[i];

    float pi_estimate = (4.0f * total_in_circle) / TOTAL_POINTS;
    printf("Estimated Pi = %f\n", pi_estimate);
    printf("Time taken: %.4f milliseconds\n", milliseconds);

    hipFree(d_count);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}

