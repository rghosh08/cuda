#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Compute distances between query and nodes
__global__ void compute_distances(float *nodes, float *query, float *distances, int dim, int num_nodes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_nodes) {
        float distance = 0;
        for (int d = 0; d < dim; d++) {
            float diff = nodes[idx * dim + d] - query[d];
            distance += diff * diff;
        }
        distances[idx] = sqrtf(distance);
    }
}

int main() {
    const int num_nodes = 1000000;
    const int dim = 128;

    float *nodes = (float*)malloc(num_nodes * dim * sizeof(float));
    float *query = (float*)malloc(dim * sizeof(float));
    float *distances = (float*)malloc(num_nodes * sizeof(float));

    // Random initialization
    for (int i = 0; i < num_nodes * dim; i++) nodes[i] = rand() / (float)RAND_MAX;
    for (int i = 0; i < dim; i++) query[i] = rand() / (float)RAND_MAX;

    float *d_nodes, *d_query, *d_distances;
    hipMalloc(&d_nodes, num_nodes * dim * sizeof(float));
    hipMalloc(&d_query, dim * sizeof(float));
    hipMalloc(&d_distances, num_nodes * sizeof(float));

    hipMemcpy(d_nodes, nodes, num_nodes * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, dim * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_nodes + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    compute_distances<<<blocksPerGrid, threadsPerBlock>>>(d_nodes, d_query, d_distances, dim, num_nodes);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(distances, d_distances, num_nodes * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print sample distances
    printf("Sample distances computed:\n");
    for (int i = 0; i < 5; i++) {
        printf("Node %d: %.4f\n", i, distances[i]);
    }

    printf("Computation Time: %.3f ms\n", milliseconds);

    hipFree(d_nodes);
    hipFree(d_query);
    hipFree(d_distances);
    free(nodes);
    free(query);
    free(distances);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
