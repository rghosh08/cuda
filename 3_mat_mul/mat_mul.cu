#include <hip/hip_runtime.h>
#include <cstdio>

#define N 1024

// Matrix multiplication kernel
__global__ void matmul(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float val = 0;
        for (int k = 0; k < width; k++) {
            val += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = val;
    }
}

int main() {
    size_t bytes = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    // Timing setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch kernel
    matmul<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    printf("Matrix multiplication completed in: %.4f milliseconds\n", milliseconds);
    printf("Sample output C[0]: %.2f\n", h_C[0]);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

