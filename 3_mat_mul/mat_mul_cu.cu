#include <hip/hip_runtime.h>
#include <cstdio>

#define N 65536

__global__ void matmul(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float val = 0;
        for (int k = 0; k < width; k++) {
            val += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = val;
    }
}

int main() {
    size_t bytes = N * N * sizeof(float);

    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Compute FLOPS
    double total_ops = 2.0 * N * N * N;
    double seconds = milliseconds / 1000.0;
    double flops = total_ops / seconds;

    printf("Execution time: %.4f ms\n", milliseconds);
    printf("Performance: %.4f GFLOPS\n", flops / 1e9);
    printf("Sample output C[0]: %.2f\n", h_C[0]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

