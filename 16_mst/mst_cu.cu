#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>

#define N 1024

__global__ void findMinEdge(int *graph, bool *inMST, int *minEdge, int *minIndex) {
    int tid = threadIdx.x;
    if (tid < N && !inMST[tid]) {
        int min = INT_MAX;
        for (int i = 0; i < N; i++) {
            if (inMST[i] && graph[i * N + tid] < min) {
                min = graph[i * N + tid];
            }
        }
        minEdge[tid] = min;
        minIndex[tid] = tid;
    } else {
        minEdge[tid] = INT_MAX;
        minIndex[tid] = -1;
    }
}

int main() {
    int graph[N * N];
    bool inMST[N] = {false};

    // Random initialization for the adjacency matrix
    for (int i = 0; i < N * N; i++) {
        graph[i] = rand() % 100 + 1;
    }

    inMST[0] = true;

    int *d_graph, *d_minEdge, *d_minIndex;
    bool *d_inMST;

    hipMalloc(&d_graph, N * N * sizeof(int));
    hipMalloc(&d_inMST, N * sizeof(bool));
    hipMalloc(&d_minEdge, N * sizeof(int));
    hipMalloc(&d_minIndex, N * sizeof(int));

    hipMemcpy(d_graph, graph, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_inMST, inMST, N * sizeof(bool), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int mst_weight = 0;

    for (int count = 1; count < N; count++) {
        findMinEdge<<<1, N>>>(d_graph, d_inMST, d_minEdge, d_minIndex);

        int minEdge[N], minIndex[N];
        hipMemcpy(minEdge, d_minEdge, N * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(minIndex, d_minIndex, N * sizeof(int), hipMemcpyDeviceToHost);

        int min_weight = INT_MAX;
        int min_node = -1;

        for (int i = 0; i < N; i++) {
            if (minEdge[i] < min_weight) {
                min_weight = minEdge[i];
                min_node = minIndex[i];
            }
        }

        mst_weight += min_weight;
        inMST[min_node] = true;

        hipMemcpy(d_inMST, inMST, N * sizeof(bool), hipMemcpyHostToDevice);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Minimum Spanning Tree Weight: " << mst_weight << "\n";
    std::cout << "Computation Time: " << milliseconds << " ms\n";


    hipFree(d_graph);
    hipFree(d_inMST);
    hipFree(d_minEdge);
    hipFree(d_minIndex);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

