
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu() {
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	printf("Hello from thread %d!\n", threadId);
}

int main() {

	hello_from_gpu<<<2, 4>>>();
	hipDeviceSynchronize();
	return 0;
}
