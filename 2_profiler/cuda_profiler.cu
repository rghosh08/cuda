#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define N 10000000
#define BLOCK_SIZE 256

__global__ void add(float *a, float *b, float *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        c[idx] = a[idx] + b[idx];
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    h_a = (float*)malloc(N*sizeof(float));
    h_b = (float*)malloc(N*sizeof(float));
    h_c = (float*)malloc(N*sizeof(float));

    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    hipMalloc(&d_a, N*sizeof(float));
    hipMalloc(&d_b, N*sizeof(float));
    hipMalloc(&d_c, N*sizeof(float));

    hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    add<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time taken (CUDA GPU): %.4f ms\n", ms);

    hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(h_a); free(h_b); free(h_c);
    return 0;
}

