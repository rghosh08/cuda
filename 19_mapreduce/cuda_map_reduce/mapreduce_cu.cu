#include "hip/hip_runtime.h"
#include "mapreduce_cu.h"

// CUDA Kernels Implementation
__global__ void map_kernel(int* input, KeyValue* mapped, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        mapped[idx].key = input[idx];
        mapped[idx].value = 1;
    }
}

__global__ void reduce_kernel(KeyValue* input, KeyValue* output, int* counts, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        int key = input[idx].key;
        atomicAdd(&counts[key], 1);
    }
}

__global__ void compact_results(int* counts, KeyValue* output, int max_keys, int* result_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < max_keys && counts[idx] > 0) {
        int pos = atomicAdd(result_count, 1);
        output[pos].key = idx;
        output[pos].value = counts[idx];
    }
}

// MapReduce Implementations
void custom_mapreduce(int* data, int n, KeyValue* results, int* num_results) {
    KeyValue *d_mapped, *d_results;
    int *d_data, *d_counts, *d_result_count;
    
    // Allocate memory
    CUDA_CHECK(hipMalloc(&d_mapped, n * sizeof(KeyValue)));
    CUDA_CHECK(hipMalloc(&d_results, MAX_WORDS * sizeof(KeyValue)));
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_counts, MAX_WORDS * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_result_count, sizeof(int)));
    
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_data, data, n * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_counts, 0, MAX_WORDS * sizeof(int)));
    CUDA_CHECK(hipMemset(d_result_count, 0, sizeof(int)));
    
    // Map phase
    int grid_size = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    map_kernel<<<grid_size, BLOCK_SIZE>>>(d_data, d_mapped, n);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Reduce phase
    reduce_kernel<<<grid_size, BLOCK_SIZE>>>(d_mapped, d_results, d_counts, n);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Compact results
    int count_grid = (MAX_WORDS + BLOCK_SIZE - 1) / BLOCK_SIZE;
    compact_results<<<count_grid, BLOCK_SIZE>>>(d_counts, d_results, MAX_WORDS, d_result_count);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy results back
    CUDA_CHECK(hipMemcpy(num_results, d_result_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(results, d_results, (*num_results) * sizeof(KeyValue), hipMemcpyDeviceToHost));
    
    // Cleanup
    hipFree(d_mapped);
    hipFree(d_results);
    hipFree(d_data);
    hipFree(d_counts);
    hipFree(d_result_count);
}

void thrust_mapreduce(int* data, int n, KeyValue* results, int* num_results) {
    thrust::device_vector<int> d_words(data, data + n);
    thrust::device_vector<int> d_counts(d_words.size(), 1);
    
    // Sort by key for grouping
    thrust::sort_by_key(d_words.begin(), d_words.end(), d_counts.begin());
    
    // Reduce by key
    thrust::device_vector<int> unique_words(n);
    thrust::device_vector<int> word_counts(n);
    
    auto end = thrust::reduce_by_key(d_words.begin(), d_words.end(), d_counts.begin(),
                                     unique_words.begin(), word_counts.begin());
    
    *num_results = end.first - unique_words.begin();
    
    // Copy results
    for (int i = 0; i < *num_results; i++) {
        results[i].key = unique_words[i];
        results[i].value = word_counts[i];
    }
}

// Test Functions
void test_word_count() {
    printf("=== WORD COUNT TEST ===\n");
    
    // Small test dataset
    int words[] = {1, 2, 1, 3, 2, 1, 4, 3, 1, 2};
    int n = sizeof(words) / sizeof(words[0]);
    
    KeyValue results[10];
    int num_results;
    
    // Test custom implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    custom_mapreduce(words, n, results, &num_results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float custom_time;
    hipEventElapsedTime(&custom_time, start, stop);
    
    printf("Custom Implementation Results:\n");
    for (int i = 0; i < num_results; i++) {
        printf("Word %d: Count %d\n", results[i].key, results[i].value);
    }
    printf("Custom Time: %.3f ms\n\n", custom_time);
    
    // Test Thrust implementation
    hipEventRecord(start);
    thrust_mapreduce(words, n, results, &num_results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float thrust_time;
    hipEventElapsedTime(&thrust_time, start, stop);
    
    printf("Thrust Implementation Results:\n");
    for (int i = 0; i < num_results; i++) {
        printf("Word %d: Count %d\n", results[i].key, results[i].value);
    }
    printf("Thrust Time: %.3f ms\n\n", thrust_time);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void test_large_dataset(long long n) {
    printf("=== LARGE DATASET PERFORMANCE TEST ===\n");
    printf("Dataset size: %lld elements\n", n);
    
    // Check memory requirements
    check_memory_requirements(n);
    
    int* data = (int*)malloc(n * sizeof(int));
    if (!data) {
        double memory_gb = (n * sizeof(int)) / (1024.0 * 1024.0 * 1024.0);
        printf("Error: Failed to allocate memory for %lld elements (%.2f GB)\n", n, memory_gb);
        return;
    }
    
    // Generate random data
    srand(time(NULL));
    long long max_unique_keys = (n < 10000) ? n : 10000;  // Limit unique keys to prevent excessive memory usage
    for (long long i = 0; i < n; i++) {
        data[i] = rand() % max_unique_keys;
    }
    
    printf("Unique keys: %lld\n", max_unique_keys);
    
    KeyValue* results = (KeyValue*)malloc(max_unique_keys * sizeof(KeyValue));
    if (!results) {
        printf("Error: Failed to allocate memory for results\n");
        free(data);
        return;
    }
    
    // Check if dataset size exceeds CUDA kernel limitations
    if (n > INT_MAX) {
        printf("Warning: Dataset size (%lld) exceeds CUDA kernel index limits.\n", n);
        printf("Processing first %d elements only.\n", INT_MAX);
        n = INT_MAX;
    }
    
    int dataset_size = (int)n;  // Safe cast after check above
    int num_results;
    
    // Performance comparison
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Test custom implementation
    printf("Running custom MapReduce...\n");
    hipEventRecord(start);
    custom_mapreduce(data, dataset_size, results, &num_results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float custom_time;
    hipEventElapsedTime(&custom_time, start, stop);
    printf("Custom MapReduce time: %.3f ms\n", custom_time);
    
    // Test Thrust implementation
    printf("Running Thrust MapReduce...\n");
    hipEventRecord(start);
    thrust_mapreduce(data, dataset_size, results, &num_results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float thrust_time;
    hipEventElapsedTime(&thrust_time, start, stop);
    printf("Thrust MapReduce time: %.3f ms\n", thrust_time);
    
    if (thrust_time > 0) {
        printf("Speedup: %.2fx\n", custom_time / thrust_time);
        printf("Throughput: %.2f M elements/sec\n", dataset_size / (thrust_time * 1000.0));
    }
    
    free(data);
    free(results);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Utility Functions
void print_usage(const char* program_name) {
    printf("Usage: %s [dataset_size]\n", program_name);
    printf("  dataset_size: Number of elements for large dataset test (default: 10000000)\n");
    printf("                Must be between 1000 and 100000000000 (100B)\n");
    printf("                Note: Large datasets require significant GPU memory\n");
    printf("\nExamples:\n");
    printf("  %s                # Run with default 10M elements\n", program_name);
    printf("  %s 1000000        # Run with 1M elements\n", program_name);
    printf("  %s 50000000       # Run with 50M elements\n", program_name);
    printf("  %s 1000000000     # Run with 1B elements (~4GB memory)\n", program_name);
    printf("  %s 10000000000    # Run with 10B elements (~40GB memory)\n", program_name);
}

void print_device_info() {
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    if (device_count == 0) {
        printf("No CUDA devices found!\n");
        exit(1);
    }
    
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("Using GPU: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Global Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Grid Size: (%d, %d, %d)\n\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void check_memory_requirements(long long n) {
    double memory_gb = (n * sizeof(int)) / (1024.0 * 1024.0 * 1024.0);
    printf("Memory required: %.2f GB\n", memory_gb);
    
    // Get available GPU memory
    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    double free_gb = free_mem / (1024.0 * 1024.0 * 1024.0);
    double total_gb = total_mem / (1024.0 * 1024.0 * 1024.0);
    
    printf("GPU Memory: %.2f GB free / %.2f GB total\n", free_gb, total_gb);
    
    if (memory_gb > free_gb * 0.8) {  // Use 80% threshold for safety
        printf("⚠️  Warning: Dataset requires %.2f GB but only %.2f GB available!\n", 
               memory_gb, free_gb);
        printf("   Consider using a smaller dataset or freeing GPU memory.\n");
    } else if (memory_gb > total_gb * 0.5) {
        printf("⚡ Info: Large dataset will use %.1f%% of GPU memory.\n", 
               (memory_gb / total_gb) * 100.0);
    }
    printf("\n");
}

// Main Function
int main(int argc, char** argv) {
    printf("CUDA MapReduce Implementation\n");
    printf("=============================\n\n");
    
    // Print device information
    print_device_info();
    
    // Parse command line arguments
    long long dataset_size = 10000000LL;  // Default 10M elements
    
    if (argc == 2) {
        if (strcmp(argv[1], "-h") == 0 || strcmp(argv[1], "--help") == 0) {
            print_usage(argv[0]);
            return 0;
        }
        
        char* endptr;
        long long parsed_size = strtoll(argv[1], &endptr, 10);
        
        if (*endptr != '\0' || parsed_size < 1000 || parsed_size > 100000000000LL) {
            printf("Error: Invalid dataset size '%s'\n", argv[1]);
            printf("Dataset size must be between 1,000 and 100,000,000,000 (100B)\n\n");
            print_usage(argv[0]);
            return 1;
        }
        
        dataset_size = parsed_size;
    } else if (argc > 2) {
        printf("Error: Too many arguments\n\n");
        print_usage(argv[0]);
        return 1;
    }
    
    // Run tests
    test_word_count();
    test_large_dataset(dataset_size);
    
    printf("MapReduce tests completed successfully!\n");
    return 0;
}
